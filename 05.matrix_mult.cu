#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define MATRIX_SIZE 25000

// CUDA kernel for matrix multiplication using global memory
__global__ void matrixMultiply(float *A, float *B, float *C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        float sum = 0.0;
        for (int k = 0; k < width; ++k) {
            sum += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = sum;
    }
}

int main() {
    int width = MATRIX_SIZE;
    size_t size = width * width * sizeof(float);

    // Host matrices and result
    float *h_A, *h_B, *h_C;
    h_A = (float *)malloc(size);
    h_B = (float *)malloc(size);
    h_C = (float *)malloc(size);

    // Initialize matrices A and B
    for (int i = 0; i < width * width; ++i) {
        h_A[i] = 1.0; // Replace with your initialization
        h_B[i] = 2.0; // Replace with your initialization
    }

    // Device matrices and result
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy matrices A and B from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block size
    int BLOCK_SIZE = 16;

    // Define grid and block dimensions
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (width + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch kernel for matrix multiplication
    matrixMultiply<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, width);

    // Copy matrix C from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify results (print some elements if needed)
    printf("Sample result: C[0][0] = %f\n", h_C[0]);

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
