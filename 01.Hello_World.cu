
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel function to print "Hello, world!" from the GPU
__global__ void helloFromGPU()
{
    printf("Hello, world from GPU\n");
}

int main()
{
    // Print "Hello, World!" from the CPU
    printf("Hello, World from CPU\n");

    // Launch kernel to print "Hello, World!" from the GPU
    helloFromGPU<<<1, 10>>>();

    // Synchronize to ensure all printf statements from the GPU are executed
    hipDeviceSynchronize();

    return 0;
}
